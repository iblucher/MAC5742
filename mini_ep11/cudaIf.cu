/*
mini EP 11

NOME: Your name here
NUSP: Your NUSP here
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

long getMS() {
    struct timespec s;
    clock_gettime(CLOCK_REALTIME, &s);
    return s.tv_sec*1000 + s.tv_nsec/1000000;
}

// number of tests
#define NTESTS 10
#define SEED 123456
#define SIZE 1024
#define WIDTH 512
#define ROUNDS 1000

long cudaSum(int *);
long cudaIfSum(int *);

int main() {
    srand(SEED);
    long cudaTime = 0;
    long cudaIfTime = 0;

    int * reference = (int *)malloc(sizeof(int)*SIZE*WIDTH);

    for(int i = 0; i < NTESTS; i++) {
        for(int j = 0; j < SIZE*WIDTH; j++) reference[j] = rand()%5096;
        cudaTime += cudaSum(reference);
        cudaIfTime += cudaIfSum(reference);
    }

    free(reference);

    printf("Average cudaTime %ldms\nAvarage cudaIfTime %ldms\n", cudaTime/NTESTS, cudaIfTime/NTESTS);
}

// Conditional Vector Sum

__global__ void cudaIfSumGPU(int *ints) {
    int sum = 0;

    int off = blockIdx.x*32 + threadIdx.x;

    for(int j = 0; j < ROUNDS; j++) {
        for(int i = 0; i < WIDTH; i++) {
            if(ints[WIDTH*off+i] % 2)
                sum += (int) sqrt((double)ints[WIDTH*off+i]);
            else
                sum += (int) sqrt((double)ints[WIDTH*off+i]);
        }

        sum = sum/128;
    }

    ints[WIDTH*SIZE+off] = sum;
}

long cudaIfSum(int *refs) {
    int *cudaRefs;
    int results[SIZE];

    hipMalloc(&cudaRefs, sizeof(int)*SIZE*(WIDTH+1));
    hipMemcpy(cudaRefs, refs, sizeof(int)*WIDTH*SIZE, hipMemcpyHostToDevice);

    long t0 = getMS();
    cudaIfSumGPU<<<32,SIZE/32>>>(cudaRefs);

    hipMemcpy(results, cudaRefs+(WIDTH*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

    int sum = 0;
    for(int i = 0; i < SIZE; i++)
        sum += results[i];

    long tf = getMS();

    hipFree(cudaRefs);

    printf("CUDA IF SUM: %d\n", sum);

    return tf-t0;
}

// Non Conditional Vector Sum

__global__ void cudaSumGPU(int *ints) {
    int sum = 0;

    int off = blockIdx.x*32 + threadIdx.x;

    for(int j = 0; j < ROUNDS; j++) {
        for(int i = 0; i < WIDTH; i++) {
            sum += (int) sqrt((double)ints[WIDTH*off+i]);
        }
        sum = sum/128;
    }

    ints[WIDTH*SIZE+off] = sum;
}

long cudaSum(int *refs) {
    int *cudaRefs;
    int results[SIZE];

    hipMalloc(&cudaRefs, sizeof(int)*SIZE*(WIDTH+1));
    hipMemcpy(cudaRefs, refs, sizeof(int)*WIDTH*SIZE, hipMemcpyHostToDevice);

    long t0 = getMS();
    cudaSumGPU<<<32,SIZE/32>>>(cudaRefs);

    hipMemcpy(results, cudaRefs+(WIDTH*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

    int sum = 0;
    for(int i = 0; i < SIZE; i++)
        sum += results[i];

    long tf = getMS();

    hipFree(cudaRefs);

    printf("CUDA SUM: %d\n", sum);

    return tf-t0;
}
