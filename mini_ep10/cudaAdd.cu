/*
mini EP 10

NOME: Isabela Blucher
NUSP: 9298170
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

long getMS() {
	struct timespec s;
	clock_gettime(CLOCK_REALTIME, &s);
	return s.tv_sec*1000 + s.tv_nsec/1000000;
}

// number of tests
#define NTESTS 10
#define SEED 123456
#define SIZE 1024

long seqSum(int *, int *);
long cudaSum(int *, int *);

int main() {
	srand(SEED);
	long seqTime = 0;
	long cudaTime = 0;

	int * reference = (int *)malloc(sizeof(int)*SIZE*SIZE);
	int result;

	for(int i = 0; i < NTESTS; i++) {
		for(int j = 0; j < SIZE*SIZE; j++) reference[j] = rand()%1024;
		seqTime += seqSum(reference, &result);
		cudaTime += cudaSum(reference, &result);
	}

	free(reference);

	printf("Average seqTime %ldms\nAvarage cudaTime %ldms\n", seqTime/NTESTS, cudaTime/NTESTS);
}

long seqSum(int *refs, int *res) {
	long t0 = getMS();

	int sum = 0;
	for(int i = 0; i < SIZE*SIZE; i++)
		sum += refs[i];
	*res = sum;

	long tf = getMS();
	return tf-t0;
}

// good tutorial link https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/

__global__ void cudaSumGPU(int *ints) {

	int index = threadIdx.x;
	int stride = blockDim.x;

	int sum = 0;
    for(int i = index; i < SIZE*SIZE; i += stride){
		sum += ints[i];
		if(i % SIZE == 0 && i > 0) {
			*(ints + (SIZE * SIZE) + (i / SIZE)) = sum;
			sum = 0;
		}
	}
	
}

long cudaSum(int *refs, int *res) {
	int *cudaRefs;
	int results[SIZE];

	// usem os ulimos 1024 ints da memoria para guardar o resultado
	hipMalloc(&cudaRefs, sizeof(int)*SIZE*(SIZE+1));
	hipMemcpy(cudaRefs, refs, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

	long t0 = getMS();

	// Experiment here
	cudaSumGPU<<< 1, SIZE >>>(cudaRefs);

	hipMemcpy(results, cudaRefs+(SIZE*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	int sum = 0;
	for(int i = 0; i < SIZE; i++) {
		sum += results[i];
	}
	long tf = getMS();

	hipFree(cudaRefs);

	puts((*res == sum)?"OK":"Err, sum mismatch");

	return tf-t0;
}
